/* This code provides an example about how to implement the two memory usage strategies.*/

#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void AplusB(int *ret, int a, int b) { ret[threadIdx.x] = a + b + threadIdx.x; }

void memoryUsage1(){
    int *ret;
    hipMallocManaged(&ret, 1000 * sizeof(int));
    AplusB<<<1, 1000>>>(ret, 10, 100);
    hipDeviceSynchronize();
    for (int i = 0; i < 1000; i++)
        printf("%d: A+B = %d\n", i, ret[i]);
    hipFree(ret);
}

void memoryUsage2() {
    int *ret;
    hipMalloc(&ret, 1000 * sizeof(int));
    AplusB<<<1, 1000>>>(ret, 10, 100);
    int *host_ret = (int *)malloc(1000 * sizeof(int));
    hipMemcpy(host_ret, ret, 1000 * sizeof(int), hipMemcpyDefault);
    for (int i = 0; i < 1000; i++)
        printf("%d: A+B = %d\n", i, host_ret[i]);
    free(host_ret);
    hipFree(ret);
}

int main() {
    memoryUsage1();
    memoryUsage2();
    return 0;
}
