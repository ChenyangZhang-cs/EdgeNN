
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N 10000000

__global__ void cuda_hello() { printf("Hello World from GPU!\n"); }

__global__ void vector_add(float *out, float *a, float *b) { out[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x]; }

int main() {
    float *a, *b, *out;
    float *d_a, *d_b, *d_out;

    a = (float *)malloc(sizeof(float) * N);
    b = (float *)malloc(sizeof(float) * N);
    out = (float *)malloc(sizeof(float) * N);
    hipMalloc((void **)&d_a, sizeof(float) * N);
    hipMalloc((void **)&d_b, sizeof(float) * N);
    hipMalloc((void **)&d_out, sizeof(float) * N);

    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);
    vector_add<<<N, 1>>>(d_out, d_a, d_b);
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    free(a);
    free(b);
    free(out);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    cuda_hello<<<1, 1>>>();
    hipDeviceSynchronize();

    return 0;
}