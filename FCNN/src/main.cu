#include "hip/hip_runtime.h"
#include "layer.cu"
#include "hip/hip_runtime.h"
#include "file.h"
#include <hip/hip_runtime.h>
#include <omp.h>
#include <sys/time.h>
#include <time.h>

using namespace std;

struct mData {
    float data[InDim];
    float *label;
    mData() { hipMallocManaged(&label, OutDim * sizeof(float)); }
};

mData train_set[train_cnt];

float testData[2] = {0.6, 0.2};
float testLabel[2] = {0.1, 0.8};

// Define layers
double mainIniTime = 0;
double iniStart = gettime();
static mLayer l_input = mLayer(0, InDim, "input");
static mLayer l_h = mLayer(InDim, hDim, "h");
static mLayer l_f = mLayer(hDim, OutDim, "output");
double iniEnd = gettime();

static double forward_propagation(float *, hipStream_t);
static void learn(hipStream_t);
static void getData();

void getData(mData *ds, char *arg = NULL) {
    int dataN;
    if (arg != NULL && strcmp(arg, "test") == 0)
        dataN = test_cnt;
    else
        dataN = train_cnt;
    float arr_in[dataN * InDim];
    readFile("./data/input.txt", arr_in, dataN * InDim);
    for (int i = 0; i < dataN; i++) {
        memcpy(ds[i].data, arr_in + i * InDim, InDim * sizeof(float));
    }
}

static void learn(hipStream_t stream1) {
    float *h_testLabel;
    hipMalloc(&h_testLabel, sizeof(float) * 2);
    hipMemcpy(h_testLabel, testLabel, sizeof(float) * 2, hipMemcpyHostToDevice);

    float err, *tmp_err;
    hipMallocManaged(&tmp_err, sizeof(float));
    int iter = 1;
    double time_taken = 0.0, total_time_taken = 0.0;
    while (iter-- > 0) {
        time_taken = 0.0;
        err = 0.0f;
        double t1 = gettime();
        for (int i = 0; i < train_cnt; ++i) {
            *tmp_err = 0;
            time_taken += forward_propagation(train_set[i].data, stream1);
        }
        double t2 = gettime();
        err /= train_cnt;
        total_time_taken += t2 - t1;
    }
    fprintf(stdout, "offset=%.1f iniTime - %lf, memcpy Time:0, malloc Time:%lf, kernel Time:%lf, ", offset,
            iniEnd - iniStart + mainIniTime, mallocEnd - mallocStart, time_taken);
}

static double forward_propagation(float *data, hipStream_t stream1) {
    for (int i = 0; i < InDim; i++)
        input_a[i] = data[i];
    l_input.clear();
    l_h.clear();
    l_f.clear();
    double start = gettime();
    if (gpurun && offset) {
        fp_z_h<<<int((hDim + 15) / 16), 16, 0, stream1>>>((float *)l_input.a, (float *)l_h.a,
                                                          (float(*)[InDim])l_h.weight, (float *)l_h.bias, offset);
    }

    if (cpurun && offset != 1)
        fp_z_h_cpu((float *)data, (float *)l_h.a, (float(*)[InDim])l_h.weight, (float *)l_h.bias, offset);

    if (offset != 1)
        hipDeviceSynchronize();

    if (gpurun && offset) {
        fp_z_f<<<int((OutDim + 15) / 16), 16, 0, stream1>>>((float *)l_h.a, l_f.a, (float(*)[hDim])l_f.weight, l_f.bias,
                                                            offset);
    }

    if (cpurun && offset != 1)
        fp_z_f_cpu((float *)l_h.a, l_f.a, (float(*)[hDim])l_f.weight, l_f.bias, offset);

    hipDeviceSynchronize();
    double end = gettime();
    return ((double)(end - start));
}

int main(int argc, const char **argv) {
    hipError_t err = hipInit(0);
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA initialisation failed with error code - %d\n", err);
        return 1;
    }
    hipStream_t stream1;
    hipStreamCreate(&stream1);
    hipStreamAttachMemAsync(stream1, &input_a, 0, hipMemAttachHost);
    hipStreamAttachMemAsync(stream1, &h_weight, 0, hipMemAttachHost);
    hipStreamAttachMemAsync(stream1, &h_bias, 0, hipMemAttachHost);
    hipStreamAttachMemAsync(stream1, &h_a, 0, hipMemAttachHost);
    hipStreamAttachMemAsync(stream1, &h_z, 0, hipMemAttachHost);
    hipStreamAttachMemAsync(stream1, &h_dweight, 0, hipMemAttachHost);
    hipStreamAttachMemAsync(stream1, &h_da, 0, hipMemAttachHost);
    hipStreamAttachMemAsync(stream1, &h_dz, 0, hipMemAttachHost);
    hipStreamAttachMemAsync(stream1, &output_weight, 0, hipMemAttachHost);
    hipStreamAttachMemAsync(stream1, &output_bias, 0, hipMemAttachHost);
    hipStreamAttachMemAsync(stream1, &output_a, 0, hipMemAttachHost);
    hipStreamAttachMemAsync(stream1, &output_z, 0, hipMemAttachHost);
    hipStreamAttachMemAsync(stream1, &output_dweight, 0, hipMemAttachHost);
    hipStreamAttachMemAsync(stream1, &output_da, 0, hipMemAttachHost);
    hipStreamAttachMemAsync(stream1, &output_dz, 0, hipMemAttachHost);
    hipDeviceSynchronize();

    getData(train_set);
    forward_propagation(train_set[0].data, stream1); // to hot up

    for (int i = 10; i >= 0; i--) {
        double total_start = gettime();
        offset = i / 10.0;
        learn(stream1);
        double total_end = gettime();
        printf("total_time:%lf\n", total_end - total_start + mainIniTime + iniEnd - iniStart);
    }
    return 0;
}

////////////////////////// device & global functions ///////////////////////////////////////

__device__ float step_function(float v) { return 1 / (1 + exp(-v)); }

__global__ void apply_step_function(float *input, float *output, const int N) {
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < N)
        output[pos] = step_function(input[pos]);
}

__global__ void makeError(float *dz, float *a, float *Y, const int N, float *err) {
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < N) {
        dz[pos] = 2 * (a[pos] - Y[pos]) * a[pos] * (1 - a[pos]) / N;
        if (err != NULL)
            atomicAdd(err, (a[pos] - Y[pos]) * (a[pos] - Y[pos]));
    }
}

__global__ void apply_grad(float *output, float *grad, const int N) {
    const int pos = blockIdx.x * blockDim.x + threadIdx.x;
    if (pos < N)
        output[pos] -= dt * grad[pos];
}

__global__ void fp_z_h(float *input, float *z, float weight[hDim][InDim], float *bias, float offset) {
    const int bIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int hDim1 = hDim * offset;
    if (bIdx < hDim1) {
        float temp = bias[bIdx];
        for (int i = 0; i < InDim; i++)
            temp += weight[bIdx][i] * input[i];
        z[bIdx] = 1 / (1 + exp(-temp));
    }
}

__global__ void fp_z_f(float *input, float *z, float weight[OutDim][hDim], float *bias, float offset) {
    const int bIdx = blockIdx.x * blockDim.x + threadIdx.x;
    int OutDim1 = OutDim * offset;
    if (bIdx < OutDim1) {
        float temp = bias[bIdx];
        for (int i = 0; i < hDim; i++)
            temp += weight[bIdx][i] * input[i];
        z[bIdx] = 1 / (1 + exp(-temp));
    }
}

/////////////////////////// corun cpu functions ///////////////////////////////////////////////
void apply_step_function_cpu(float *input, float *output, const int N) {
    const int startN = N * offset;

    for (int idx = startN; idx < N; ++idx) {
        output[idx] = sigmoid(input[idx]);
    }
}

void makeError_cpu(float *dz, float *a, float *Y, const int N, float *err) {
    const int startN = N * 0;
    for (int pos = startN; pos < N; pos++) {
        dz[pos] = 2 * (a[pos] - Y[pos]) * a[pos] * (1 - a[pos]) / N;
        if (err != NULL)
            (*err) += (a[pos] - Y[pos]) * (a[pos] - Y[pos]);
    }
}

void apply_grad_cpu(float *output, float *grad, const int N) {
    const int startN = N * offset;
    for (int idx = startN; idx < N; ++idx) {
        output[idx] -= dt * grad[idx];
    }
}

void fp_z_h_cpu(float *input, float *z, float weight[hDim][InDim], float *bias, float offset) {
    const int hDim1 = hDim * offset;
#pragma omp parallel for
    for (int bIdx = hDim1; bIdx < hDim; bIdx++) {
        float temp = bias[bIdx];
        for (int idx = 0; idx < InDim; ++idx) {
            temp += weight[bIdx][idx] * input[idx];
        }
        z[bIdx] = 1 / (1 + exp(-temp));
    }
}

void fp_z_f_cpu(float *input, float *z, float weight[OutDim][hDim], float *bias, float offset) {
    const int OutDim1 = OutDim * offset;
#pragma omp parallel for
    for (int bIdx = OutDim1; bIdx < OutDim; bIdx++) {
        float temp = bias[bIdx];
        for (int idx = 0; idx < hDim; ++idx) {
            temp += weight[bIdx][idx] * input[idx];
        }
        z[bIdx] = 1 / (1 + exp(-temp));
    }
}