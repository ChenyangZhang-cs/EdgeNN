#include "hip/hip_runtime.h"
#define USE_MNIST_LOADER
#define MNIST_DOUBLE
#include "layer.cu"
#include "mnist.h"
#include <cstdio>
#include <hip/hip_runtime.h>
#include <time.h>

static mnist_data *train_set, *test_set;
static unsigned int train_cnt, test_cnt;

// Define layers of CNN
double iniStart = gettime();
static Layer l_input = Layer(0, 0, 28 * 28, "input");
static Layer l_c1 = Layer(5 * 5, 6, 24 * 24 * 6, "c1");
static Layer l_c2 = Layer(2 * 2, 6, 12 * 12 * 6, "c2");
static Layer l_c3 = Layer(2 * 2, 6, 6 * 6 * 6, "c3");
static Layer l_f = Layer(6 * 6 * 6, 10, 10, "f");
static Layer l_r = Layer(4 * 4, 1, 6 * 6 * 6, "r");
double iniEnd = gettime();
static void learn();
static double forward_pass(double data[28][28]);

static inline void loaddata() {
    mnist_load("data/train-images.idx3-ubyte", "data/train-labels.idx1-ubyte", &train_set, &train_cnt);
    mnist_load("data/t10k-images.idx3-ubyte", "data/t10k-labels.idx1-ubyte", &test_set, &test_cnt);
}

inline void get_cuda_size(const int N, int &grid, int &block) {
    int i = -1;
    int temp = N;
    while (temp) {
        temp >>= 1;
        i++;
    }
    block = 1 << int(i / 2);
    grid = ceil(1.0 * N / block);
}

int main(int argc, const char **argv) {
    srand(time(NULL));
    hipError_t err = hipInit(0);
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA initialisation failed with error code - %d\n", err);
        return 1;
    }
    //////////////////////////////cudastream////////////////////////////////////////
    hipStream_t stream1;
    hipStreamCreate(&stream1);
    hipStreamAttachMemAsync(stream1, &input_a, 0, hipMemAttachHost);
    hipStreamAttachMemAsync(stream1, &c1_weight, 0, hipMemAttachHost);
    hipStreamAttachMemAsync(stream1, &c1_bias, 0, hipMemAttachHost);
    hipStreamAttachMemAsync(stream1, &c1_a, 0, hipMemAttachHost);
    hipStreamAttachMemAsync(stream1, &c1_z, 0, hipMemAttachHost);
    hipStreamAttachMemAsync(stream1, &c2_weight, 0, hipMemAttachHost);
    hipStreamAttachMemAsync(stream1, &c2_bias, 0, hipMemAttachHost);
    hipStreamAttachMemAsync(stream1, &c2_a, 0, hipMemAttachHost);
    hipStreamAttachMemAsync(stream1, &c2_z, 0, hipMemAttachHost);
    hipStreamAttachMemAsync(stream1, &c3_weight, 0, hipMemAttachHost);
    hipStreamAttachMemAsync(stream1, &c3_bias, 0, hipMemAttachHost);
    hipStreamAttachMemAsync(stream1, &c3_a, 0, hipMemAttachHost);
    hipStreamAttachMemAsync(stream1, &c3_z, 0, hipMemAttachHost);
    hipStreamAttachMemAsync(stream1, &f_weight, 0, hipMemAttachHost);
    hipStreamAttachMemAsync(stream1, &f_bias, 0, hipMemAttachHost);
    hipStreamAttachMemAsync(stream1, &f_a, 0, hipMemAttachHost);
    hipStreamAttachMemAsync(stream1, &f_z, 0, hipMemAttachHost);
    hipStreamAttachMemAsync(stream1, &r_weight, 0, hipMemAttachHost);
    hipStreamAttachMemAsync(stream1, &r_bias, 0, hipMemAttachHost);
    hipStreamAttachMemAsync(stream1, &r_z, 0, hipMemAttachHost);

    loaddata();
    for (int i = 10; i >= 0; i--) {
        double totalSt = gettime();
        offset = i / 10.0;
        learn();
        double totalEnd = gettime();
        printf("Total Time:%lf\n", totalEnd - totalSt);
    }
    return 0;
}

// Forward propagation of a single row in dataset

static double forward_pass(double data[28][28]) {
    for (int i = 0; i < 28; ++i) {
        for (int j = 0; j < 28; ++j) {
            input_a[i * 28 + j] = data[i][j];
        }
    }
    double start = gettime();

    if (offset) {
        int c1_grid, c1_block;
        get_cuda_size(int(6 * 24 * 24 * offset), c1_grid, c1_block);
        fp_preact_c1<<<c1_grid, c1_block>>>((float(*)[28])l_input.output, (float(*)[24][24])l_c1.output,
                                            (float(*)[5][5])l_c1.weight, l_c1.bias, offset);
    }
    if (offset != 1)
        fp_preact_c1_cpu((float(*)[28])l_input.output, (float(*)[24][24])l_c1.output, (float(*)[5][5])l_c1.weight,
                         l_c1.bias);

    if (offset != 1)
        hipDeviceSynchronize();

    if (offset) {
        int r_grid, r_block;
        get_cuda_size(int(6 * 6 * 6 * offset), r_grid, r_block);
        fp_preact_r<<<r_grid, r_block>>>((float(*)[24][24])l_c1.output, (float(*)[6][6])l_r.preact,
                                         (float(*)[4][4])l_r.weight, *l_r.bias, offset);
    }
    if (offset != 1)
        fp_preact_r_cpu((float(*)[24][24])l_c1.output, (float(*)[6][6])l_r.preact, (float(*)[4][4])l_r.weight,
                        *l_r.bias);
    if (offset != 1)
        hipDeviceSynchronize();

    if (offset) {
        int c2_grid, c2_block;
        get_cuda_size(int(6 * 12 * 12 * offset), c2_grid, c2_block);
        fp_preact_c2<<<c2_grid, c2_block>>>((float(*)[24][24])l_c1.output, (float(*)[12][12])l_c2.output,
                                            (float(*)[2][2])l_c2.weight, l_c2.bias, offset);
    }
    if (offset != 1)
        fp_preact_c2_cpu((float(*)[24][24])l_c1.output, (float(*)[12][12])l_c2.output, (float(*)[2][2])l_c2.weight,
                         l_c2.bias);

    if (offset != 1)
        hipDeviceSynchronize();

    if (offset) {
        int c3_grid, c3_block;
        get_cuda_size(int(6 * 6 * 6 * offset), c3_grid, c3_block);
        fp_preact_c3<<<c3_grid, c3_block>>>((float(*)[12][12])l_c2.output, (float(*)[6][6])l_c3.preact,
                                            (float(*)[2][2])l_c3.weight, l_c3.bias, offset);
    }
    if (offset != 1)
        fp_preact_c3_cpu((float(*)[12][12])l_c2.output, (float(*)[6][6])l_c3.preact, (float(*)[2][2])l_c3.weight,
                         l_c3.bias);

    int add_grid, add_block;
    get_cuda_size(6 * 6 * 6, add_grid, add_block);
    fp_add_res<<<add_grid, add_block>>>((float(*)[6][6])l_c3.preact, (float(*)[6][6])l_r.preact);
    apply_sigmoid<<<128, 128>>>(l_c3.preact, l_c3.output, l_c3.O);

    if (offset != 1)
        hipDeviceSynchronize();

    if (offset) {
        int f_grid, f_block;
        get_cuda_size(int(10 * offset), f_grid, f_block);
        fp_preact_f<<<f_grid, f_block>>>((float(*)[6][6])l_c3.output, l_f.output, (float(*)[6][6][6])l_f.weight,
                                         l_f.bias, offset);
    }
    if (offset != 1)
        fp_preact_f_cpu((float(*)[6][6])l_c3.output, l_f.output, (float(*)[6][6][6])l_f.weight, l_f.bias);

    hipDeviceSynchronize();

    double end = gettime();

    return ((double)(end - start));
}

static void unfold_input(double input[28][28], double unfolded[24 * 24][5 * 5]) {
    int a = 0;
    (void)unfold_input;
    for (int i = 0; i < 2; ++i)
        for (int j = 0; j < 2; ++j) {
            int b = 0;
            for (int x = i; x < i + 2; ++x)
                for (int y = j; y < j + 2; ++y)
                    unfolded[a][b++] = input[x][y];
            a++;
        }
}

static void learn() {
    int iter = 1;
    double time_taken = 0.0;
    while (iter < 0 || iter-- > 0) {
        for (int i = 0; i < 500; ++i) {
            time_taken += forward_pass(train_set[i].data);
        }
    }
    fprintf(stdout, "offset=%.1f iniTime - %lf, memcpy Time:0, malloc Time:%lf, kernel Time:%lf,", offset,
            iniEnd - iniStart, mallocEnd - mallocStart, time_taken);
}
